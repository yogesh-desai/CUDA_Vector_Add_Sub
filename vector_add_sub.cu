#include "hip/hip_runtime.h"
/*Title: Vector addition and subtraction in CUDA.
A simple way to understand how CUDA can be used to perform arithmetic operations.
*/
#include<iostream>
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>
using namespace std;
# define size 5

//Global functions
__global__ void AddIntsCUDA(int *a, int *b)
{
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	a[tid] = a[tid] + b[tid];
}

__global__ void SubIntsCUDA(int *a, int *b)
{
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	b[tid] = a[tid] - b[tid];
}
//********************************************************
int main()
{
	int a[size]={1,2,3,4,5}, b[size]={1,2,3,4,5}; //Vector Declaration and Definition
	int *d_a, *d_b;

	//Allocation of Device variables
	hipMalloc((void **)&d_a, sizeof(int)*size);
	hipMalloc((void **)&d_b, sizeof(int)*size);

	//Copy Host Memory to Device Memory
	hipMemcpy(d_a, &a, sizeof(int)*size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, sizeof(int)*size, hipMemcpyHostToDevice);
	
	//Launch Kernel
	AddIntsCUDA << <2,3 >> >(d_a, d_b);
	
	//Copy Device Memory to Host Memory
	hipMemcpy(&a, d_a, sizeof(int)*size, hipMemcpyDeviceToHost);

	cout << "The answer is "<<endl;
	for(int i=0;i<size;i++)
	{
		printf("a[%d]=%d\n",i,a[i]);
	}

	//Deallocate the Device Memory and Host Memory
	hipFree(d_a);
	hipFree(d_b);
	free(a);
	free(b);

	return 0;
}
