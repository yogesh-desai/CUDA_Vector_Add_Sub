#include "hip/hip_runtime.h"

#include<iostream>
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>
using namespace std;
# define size 5
__global__ void AddIntsCUDA(int *a, int *b)
{
	//for(int i=0;i<size;i++)
	//{
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	a[tid] = a[tid] + b[tid];

	//}
}
__global__ void SubIntsCUDA(int *a, int *b)

{
	//for(int i=0;i<size;i++)
	//{
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	b[tid] = a[tid] - b[tid];

	//}
}

int main()
{
	int a[size]={1,2,3,4,5}, b[size]={1,2,3,4,5};
	int *d_a, *d_b;

	hipMalloc((void **)&d_a, sizeof(int)*size);
	hipMalloc((void **)&d_b, sizeof(int)*size);
	hipMemcpy(d_a, &a, sizeof(int)*size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, sizeof(int)*size, hipMemcpyHostToDevice);

	AddIntsCUDA << <2,3 >> >(d_a, d_b);

	hipMemcpy(&a, d_a, sizeof(int)*size, hipMemcpyDeviceToHost);


	cout << "The answer is "<<endl;
	for(int i=0;i<size;i++)
	{
		printf("a[%d]=%d\n",i,a[i]);
	}
	hipFree(d_a);
	hipFree(d_b);
	free(a);
	free(b);

	/*int a[size]={1,2,3,4,5}, b[size]={1,2,3,4,5};

	hipMalloc((void **)&d_a, sizeof(int)*size);
	hipMalloc((void **)&d_b, sizeof(int)*size);
	hipMemcpy(d_a, &a, sizeof(int)*size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, sizeof(int)*size, hipMemcpyHostToDevice);

	SubIntsCUDA << <2,3 >> >(d_a, d_b);
	hipMemcpy(&b, d_b, sizeof(int)*size, hipMemcpyDeviceToHost);
	cout << "\nThe Subtraction is "<<endl;
		for(int i=0;i<size;i++)
		{
			printf("b[%d]=%d\n",i,b[i]);
		}
*
	hipFree(d_a);
	hipFree(d_b);

*/

//	AddIntsCUDA << <1, 1 >> >(d_a, d_b);


	return 0;
}

